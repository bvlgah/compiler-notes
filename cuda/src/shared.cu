#include "hip/hip_runtime.h"
#include <algorithm>
#include <array>
#include <cstdint>
#include <cstddef>
#include <cstdlib>
#include <iostream>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

__global__ void sharedMemoryReadKernel(
    std::int32_t * __restrict__ output, const std::size_t stride) {
  extern __shared__ std::uint32_t shared[];
  const std::size_t tid = threadIdx.x + blockIdx.x * blockDim.x;
  const std::size_t idx = tid * stride;

  auto startClock = clock();
  shared[idx]++;
  auto endClock = clock();
  output[tid] = endClock - startClock;
}

static std::ostream
&operator<<(std::ostream &output,
    const thrust::host_vector<std::int32_t> &vec) {
  auto it = vec.begin();
  output << '[';

  if (it != vec.end()) {
    output << *it;
    while (++it != vec.end())
      output << ", " << *it;
  }

  output << ']';
  return output;
}

void sharedMemoryThroughput(std::size_t stride, std::size_t threadSize) {
  const std::size_t sharedSize = threadSize * stride * sizeof(std::uint32_t);
  thrust::device_vector<std::int32_t> output(threadSize);
  sharedMemoryReadKernel<<<1, threadSize, sharedSize>>>(
      thrust::raw_pointer_cast(output.data()), stride);

  thrust::host_vector<std::int32_t> timingVec = output;
  std::int32_t maxTiming = *std::max_element(
      timingVec.begin(), timingVec.end());
  std::int32_t minTiming = *std::min_element(
      timingVec.begin(), timingVec.end());
  std::cout << "Stride is " << stride
            << ", ThreadSize is " << threadSize << std::endl;
  std::cout << "Timing vector is " << timingVec << std::endl;
  std::cout << "Max Timing is " << maxTiming << " clocks" << std::endl;
  std::cout << "Min Timing is " << minTiming << " clocks" << std::endl;
}

int main(void) {
  std::array<std::size_t, 8> strides {1, 2, 4, 8, 16, 28, 31, 32};
  for (std::size_t stride : strides)
    sharedMemoryThroughput(stride, 32);
  return EXIT_SUCCESS;
}
